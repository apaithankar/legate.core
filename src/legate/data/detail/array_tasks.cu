/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "legate/cuda/hip/hip_runtime.h"
#include "legate/data/detail/array_tasks.h"
#include "legate/task/task_context.h"

namespace legate::detail {

namespace {

LEGATE_DEVICE inline std::size_t global_tid_1d()
{
  return static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
}

template <typename DescAcc>
LEGATE_KERNEL void fixup_ranges(std::size_t desc_volume,
                                Point<1> desc_lo,
                                Point<1> vardata_lo,
                                DescAcc desc_acc)
{
  auto tid = global_tid_1d();
  if (tid >= desc_volume) {
    return;
  }
  auto& desc = desc_acc[desc_lo + tid];
  desc.lo += vardata_lo;
  desc.hi += vardata_lo;
}

}  // namespace

/*static*/ void FixupRanges::gpu_variant(legate::TaskContext context)
{
  if (context.get_task_index()[0] == 0) {
    return;
  }

  // TODO(wonchanl): We need to extend this to nested cases
  const auto num_outputs = context.num_outputs();
  auto stream            = context.get_task_stream();

  for (std::uint32_t i = 0; i < num_outputs; ++i) {
    const auto list_arr   = context.output(i).as_list_array();
    const auto desc       = list_arr.descriptor();
    const auto desc_shape = desc.shape<1>();
    if (desc_shape.empty()) {
      continue;
    }

    auto vardata_lo = list_arr.vardata().shape<1>().lo;
    auto desc_acc   = desc.data().read_write_accessor<Rect<1>, 1>();

    std::size_t desc_volume = desc_shape.volume();
    auto num_blocks = (desc_volume + LEGATE_THREADS_PER_BLOCK - 1) / LEGATE_THREADS_PER_BLOCK;
    fixup_ranges<<<num_blocks, LEGATE_THREADS_PER_BLOCK, 0, stream>>>(
      desc_volume, desc_shape.lo, vardata_lo, desc_acc);
  }
}
namespace {

template <typename RangesAcc, typename OffsetsAcc>
LEGATE_KERNEL void offsets_to_ranges(std::size_t offsets_volume,
                                     std::int64_t vardata_volume,
                                     Point<1> offsets_lo,
                                     Point<1> vardata_lo,
                                     RangesAcc ranges_acc,
                                     OffsetsAcc offsets_acc)
{
  auto tid = global_tid_1d();
  if (tid >= offsets_volume) {
    return;
  }
  auto p      = offsets_lo + tid;
  auto& range = ranges_acc[p];
  range.lo[0] = vardata_lo + offsets_acc[p];
  range.hi[0] = vardata_lo + (tid != offsets_volume - 1 ? offsets_acc[p + 1] : vardata_volume) - 1;
}

}  // namespace

/*static*/ void OffsetsToRanges::gpu_variant(legate::TaskContext context)
{
  auto offsets = context.input(0).data();
  auto vardata = context.input(1).data();
  auto ranges  = context.output(0).data();

  auto offsets_shape = offsets.shape<1>();
  LEGATE_CHECK(offsets_shape == ranges.shape<1>());

  if (offsets_shape.empty()) {
    return;
  }

  auto vardata_shape = vardata.shape<1>();
  auto vardata_lo    = vardata_shape.lo[0];

  auto offsets_acc = offsets.read_accessor<int32_t, 1>();
  auto ranges_acc  = ranges.write_accessor<Rect<1>, 1>();

  auto stream = context.get_task_stream();

  std::size_t offsets_volume = offsets_shape.volume();
  std::size_t vardata_volume = vardata_shape.volume();

  auto num_blocks = (offsets_volume + LEGATE_THREADS_PER_BLOCK - 1) / LEGATE_THREADS_PER_BLOCK;
  offsets_to_ranges<<<num_blocks, LEGATE_THREADS_PER_BLOCK, 0, stream>>>(
    offsets_volume, vardata_volume, offsets_shape.lo, vardata_shape.lo, ranges_acc, offsets_acc);
}

namespace {

template <typename OffsetsAcc, typename RangesAcc>
LEGATE_KERNEL void ranges_to_offsets(std::size_t ranges_volume,
                                     Point<1> ranges_lo,
                                     OffsetsAcc offsets_acc,
                                     RangesAcc ranges_acc)
{
  auto tid = global_tid_1d();
  if (tid >= ranges_volume) {
    return;
  }
  auto p         = ranges_lo + tid;
  offsets_acc[p] = ranges_acc[p].lo[0] - ranges_acc[ranges_lo].lo[0];
}

}  // namespace

/*static*/ void RangesToOffsets::gpu_variant(legate::TaskContext context)
{
  auto ranges  = context.input(0).data();
  auto offsets = context.output(0).data();

  auto ranges_shape = ranges.shape<1>();
  LEGATE_CHECK(ranges_shape == offsets.shape<1>());

  if (ranges_shape.empty()) {
    return;
  }

  auto ranges_acc  = ranges.read_accessor<Rect<1>, 1>();
  auto offsets_acc = offsets.write_accessor<int32_t, 1>();

  auto stream = context.get_task_stream();

  auto ranges_volume = ranges_shape.volume();
  auto num_blocks    = (ranges_volume + LEGATE_THREADS_PER_BLOCK - 1) / LEGATE_THREADS_PER_BLOCK;
  ranges_to_offsets<<<num_blocks, LEGATE_THREADS_PER_BLOCK, 0, stream>>>(
    ranges_volume, ranges_shape.lo, offsets_acc, ranges_acc);
}

}  // namespace legate::detail
