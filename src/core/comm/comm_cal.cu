/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "core/comm/coll.h"
#include "core/comm/comm_cal.h"
#include "core/cuda/cuda_help.h"
#include "core/cuda/stream_pool.h"
#include "core/data/buffer.h"
#include "core/operation/detail/task_launcher.h"
#include "core/runtime/detail/communicator_manager.h"
#include "core/runtime/detail/library.h"
#include "core/runtime/detail/runtime.h"
#include "core/runtime/runtime.h"
#include "core/task/detail/task_context.h"
#include "core/utilities/nvtx_help.h"
#include "core/utilities/typedefs.h"

#include <cal.h>
#include <chrono>
#include <hip/hip_runtime.h>

namespace legate::detail {

void show_progress(const Legion::Task* task, Legion::Context ctx, Legion::Runtime* runtime);

}  // namespace legate::detail

namespace legate::comm::cal {

#define CHECK_CAL(...)                     \
  do {                                     \
    const calError_t result = __VA_ARGS__; \
    check_cal(result, __FILE__, __LINE__); \
  } while (false)

namespace {

const char* cal_get_error_message(calError_t error)
{
  switch (error) {
    case CAL_OK: return "Success";
    case CAL_ERROR_INPROGRESS: return "Request is in progress";
    case CAL_ERROR: return "Generic error";
    case CAL_ERROR_INVALID_PARAMETER: return "Invalid parameter to the interface function";
    case CAL_ERROR_INTERNAL: return "Internal error";
    case CAL_ERROR_CUDA: return "Error in CUDA runtime/driver API";
    case CAL_ERROR_UCC: return "Error in UCC call";
    case CAL_ERROR_NOT_SUPPORTED: return "Requested configuration or parameters are not supported";
    default: return "Unknown error code";
  }
}

void check_cal(calError_t error, const char* file, int line)
{
  if (error != CAL_OK) {
    static_cast<void>(fprintf(stderr,
                              "Internal CAL failure with error %d (%s) in file %s at line %d\n",
                              error,
                              cal_get_error_message(error),
                              file,
                              line));
    exit(error);
  }
}

}  // namespace

class Factory final : public detail::CommunicatorFactory {
 public:
  explicit Factory(const detail::Library* core_library);

 public:
  [[nodiscard]] bool needs_barrier() const override;
  [[nodiscard]] bool is_supported_target(mapping::TaskTarget target) const override;

 protected:
  [[nodiscard]] Legion::FutureMap initialize(const mapping::detail::Machine& machine,
                                             uint32_t num_tasks) override;
  void finalize(const mapping::detail::Machine& machine,
                uint32_t num_tasks,
                const Legion::FutureMap& communicator) override;

 private:
  const detail::Library* core_library_{};
};

Factory::Factory(const detail::Library* core_library) : core_library_{core_library} {}

bool Factory::needs_barrier() const { return true; }

bool Factory::is_supported_target(mapping::TaskTarget target) const
{
  return target == mapping::TaskTarget::GPU;
}

Legion::FutureMap Factory::initialize(const mapping::detail::Machine& machine, uint32_t num_tasks)
{
  Domain launch_domain{Rect<1>{Point<1>{0}, Point<1>{static_cast<int64_t>(num_tasks) - 1}}};

  detail::TaskLauncher init_cal_launcher{
    core_library_, machine, LEGATE_CORE_INIT_CAL_TASK_ID, LEGATE_GPU_VARIANT};
  init_cal_launcher.set_concurrent(true);

  // add cpu communicator
  auto* comm_mgr         = detail::Runtime::get_runtime()->communicator_manager();
  auto* cpu_comm_factory = comm_mgr->find_factory("cpu");
  auto cpu_comm          = cpu_comm_factory->find_or_create(
    mapping::TaskTarget::GPU, machine.processor_range(), launch_domain);
  init_cal_launcher.add_future_map(cpu_comm);

  return init_cal_launcher.execute(launch_domain);
}

void Factory::finalize(const mapping::detail::Machine& machine,
                       uint32_t num_tasks,
                       const Legion::FutureMap& communicator)
{
  Domain launch_domain{Rect<1>{Point<1>{0}, Point<1>{static_cast<int64_t>(num_tasks) - 1}}};

  detail::TaskLauncher launcher{
    core_library_, machine, LEGATE_CORE_FINALIZE_CAL_TASK_ID, LEGATE_GPU_VARIANT};
  launcher.set_concurrent(true);
  launcher.add_future_map(communicator);
  launcher.execute(launch_domain);
}

namespace {

[[nodiscard]] calError_t allgather(
  void* src_buf, void* recv_buf, size_t size, void* data, void** request)
{
  // this is sync!
  auto res = comm::coll::collAllgather(src_buf,
                                       recv_buf,
                                       size,
                                       comm::coll::CollDataType::CollInt8,
                                       reinterpret_cast<comm::coll::CollComm>(data));
  if (res != 0) {
    return CAL_ERROR;
  }

  // some dummy request
  auto dummy = new calError_t{};
  *request   = static_cast<void*>(dummy);

  return CAL_OK;
}

[[nodiscard]] calError_t request_test(void*) { return CAL_OK; }
[[nodiscard]] calError_t request_free(void* request)
{
  delete reinterpret_cast<calError_t*>(request);
  return CAL_OK;
}

[[nodiscard]] cal_comm_t init_cal(const Legion::Task* task,
                                  const std::vector<Legion::PhysicalRegion>& /*regions*/,
                                  Legion::Context context,
                                  Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range{"core::comm::cal::init"};

  legate::detail::show_progress(task, context, runtime);

  auto rank      = task->index_point[0];
  auto num_ranks = task->index_domain.get_volume();

  LegateCheck(task->futures.size() == 1);
  auto cpu_comm = task->futures[0].get_result<comm::coll::CollComm>();

  int device = -1;
  CHECK_CUDA(hipGetDevice(&device));

  /* Create communicator */
  cal_comm_t cal_comm = nullptr;
  cal_comm_create_params_t params;
  params.allgather    = allgather;
  params.req_test     = request_test;
  params.req_free     = request_free;
  params.data         = reinterpret_cast<void*>(cpu_comm);
  params.rank         = rank;
  params.nranks       = num_ranks;
  params.local_device = device;

  CHECK_CAL(cal_comm_create(params, &cal_comm));

  return cal_comm;
}

void finalize_cal(const Legion::Task* task,
                  const std::vector<Legion::PhysicalRegion>& /*regions*/,
                  Legion::Context context,
                  Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range{"core::comm::cal::finalize"};

  legate::detail::show_progress(task, context, runtime);

  LegateCheck(task->futures.size() == 1);
  auto comm = task->futures[0].get_result<cal_comm_t>();
  CHECK_CAL(cal_comm_destroy(comm));
}

}  // namespace

void register_tasks(const detail::Library* core_library)
{
  const auto runtime    = Legion::Runtime::get_runtime();
  auto init_cal_task_id = core_library->get_task_id(LEGATE_CORE_INIT_CAL_TASK_ID);
  constexpr const char* init_cal_task_name = "core::comm::cal::init";
  runtime->attach_name(
    init_cal_task_id, init_cal_task_name, false /*mutable*/, true /*local only*/);

  auto finalize_cal_task_id = core_library->get_task_id(LEGATE_CORE_FINALIZE_CAL_TASK_ID);
  constexpr const char* finalize_cal_task_name = "core::comm::cal::finalize";
  runtime->attach_name(
    finalize_cal_task_id, finalize_cal_task_name, false /*mutable*/, true /*local only*/);

  auto make_registrar = [&](auto task_id, auto* task_name, auto proc_kind) {
    Legion::TaskVariantRegistrar registrar{task_id, task_name};

    registrar.add_constraint(Legion::ProcessorConstraint{proc_kind});
    registrar.set_leaf(true);
    registrar.global_registration = false;
    return registrar;
  };

  // Register the task variants
  {
    auto registrar = make_registrar(init_cal_task_id, init_cal_task_name, Processor::TOC_PROC);
    runtime->register_task_variant<cal_comm_t, init_cal>(registrar, LEGATE_GPU_VARIANT);
  }
  {
    auto registrar =
      make_registrar(finalize_cal_task_id, finalize_cal_task_name, Processor::TOC_PROC);
    runtime->register_task_variant<finalize_cal>(registrar, LEGATE_GPU_VARIANT);
  }
}

void register_factory(const detail::Library* core_library)
{
  auto* comm_mgr = detail::Runtime::get_runtime()->communicator_manager();
  comm_mgr->register_factory("cal", std::make_unique<Factory>(core_library));
}

}  // namespace legate::comm::cal
