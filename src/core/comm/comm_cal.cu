/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "core/comm/coll.h"
#include "core/comm/comm_cal.h"
#include "core/comm/comm_util.h"
#include "core/cuda/hip/hip_runtime.h"
#include "core/data/buffer.h"
#include "core/operation/detail/task_launcher.h"
#include "core/runtime/detail/communicator_manager.h"
#include "core/runtime/detail/library.h"
#include "core/runtime/detail/runtime.h"
#include "core/runtime/runtime.h"
#include "core/utilities/assert.h"
#include "core/utilities/nvtx_help.h"
#include "core/utilities/typedefs.h"

#include <cal.h>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <memory>
#include <vector>

namespace legate::detail {

void show_progress(const Legion::Task* task, Legion::Context ctx, Legion::Runtime* runtime);

}  // namespace legate::detail

namespace legate::comm::cal {

#define CHECK_CAL(...)                     \
  do {                                     \
    const calError_t result = __VA_ARGS__; \
    check_cal(result, __FILE__, __LINE__); \
  } while (false)

namespace {

const char* cal_get_error_message(calError_t error)
{
  switch (error) {
    case CAL_OK: return "Success";
    case CAL_ERROR_INPROGRESS: return "Request is in progress";
    case CAL_ERROR: return "Generic error";
    case CAL_ERROR_INVALID_PARAMETER: return "Invalid parameter to the interface function";
    case CAL_ERROR_INTERNAL: return "Internal error";
    case CAL_ERROR_CUDA: return "Error in CUDA runtime/driver API";
    case CAL_ERROR_UCC: return "Error in UCC call";
    case CAL_ERROR_NOT_SUPPORTED: return "Requested configuration or parameters are not supported";
    default: return "Unknown error code";
  }
}

void check_cal(calError_t error, const char* file, int line)
{
  if (error != CAL_OK) {
    static_cast<void>(fprintf(stderr,
                              "Internal CAL failure with error %d (%s) in file %s at line %d\n",
                              error,
                              cal_get_error_message(error),
                              file,
                              line));
    std::exit(error);
  }
}

}  // namespace

class Factory final : public detail::CommunicatorFactory {
 public:
  explicit Factory(const detail::Library* core_library);

 public:
  [[nodiscard]] bool needs_barrier() const override;
  [[nodiscard]] bool is_supported_target(mapping::TaskTarget target) const override;

 protected:
  [[nodiscard]] Legion::FutureMap initialize(const mapping::detail::Machine& machine,
                                             std::uint32_t num_tasks) override;
  void finalize(const mapping::detail::Machine& machine,
                std::uint32_t num_tasks,
                const Legion::FutureMap& communicator) override;

 private:
  const detail::Library* core_library_{};
};

Factory::Factory(const detail::Library* core_library) : core_library_{core_library} {}

bool Factory::needs_barrier() const { return true; }

bool Factory::is_supported_target(mapping::TaskTarget target) const
{
  return target == mapping::TaskTarget::GPU;
}

Legion::FutureMap Factory::initialize(const mapping::detail::Machine& machine,
                                      std::uint32_t num_tasks)
{
  Domain launch_domain{Rect<1>{Point<1>{0}, Point<1>{static_cast<std::int64_t>(num_tasks) - 1}}};

  detail::TaskLauncher init_cal_launcher{
    core_library_, machine, LEGATE_CORE_INIT_CAL_TASK_ID, LEGATE_GPU_VARIANT};
  init_cal_launcher.set_concurrent(true);

  // add cpu communicator
  auto* comm_mgr         = detail::Runtime::get_runtime()->communicator_manager();
  auto* cpu_comm_factory = comm_mgr->find_factory("cpu");
  auto cpu_comm          = cpu_comm_factory->find_or_create(
    mapping::TaskTarget::GPU, machine.processor_range(), launch_domain);
  init_cal_launcher.add_future_map(cpu_comm);

  return init_cal_launcher.execute(launch_domain);
}

void Factory::finalize(const mapping::detail::Machine& machine,
                       std::uint32_t num_tasks,
                       const Legion::FutureMap& communicator)
{
  Domain launch_domain{Rect<1>{Point<1>{0}, Point<1>{static_cast<std::int64_t>(num_tasks) - 1}}};

  detail::TaskLauncher launcher{
    core_library_, machine, LEGATE_CORE_FINALIZE_CAL_TASK_ID, LEGATE_GPU_VARIANT};
  launcher.set_concurrent(true);
  launcher.add_future_map(communicator);
  launcher.execute(launch_domain);
}

namespace {

[[nodiscard]] calError_t allgather(
  void* src_buf, void* recv_buf, std::size_t size, void* data, void** request)
{
  // this is sync!
  auto res = comm::coll::collAllgather(src_buf,
                                       recv_buf,
                                       size,
                                       comm::coll::CollDataType::CollInt8,
                                       reinterpret_cast<comm::coll::CollComm>(data));
  if (res != 0) {
    return CAL_ERROR;
  }

  // some dummy request
  auto dummy = new calError_t{};
  *request   = static_cast<void*>(dummy);

  return CAL_OK;
}

[[nodiscard]] calError_t request_test(void*) { return CAL_OK; }
[[nodiscard]] calError_t request_free(void* request)
{
  delete reinterpret_cast<calError_t*>(request);
  return CAL_OK;
}

[[nodiscard]] cal_comm_t init_cal(const Legion::Task* task,
                                  const std::vector<Legion::PhysicalRegion>& /*regions*/,
                                  Legion::Context context,
                                  Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range{"core::comm::cal::init"};

  legate::detail::show_progress(task, context, runtime);

  auto rank      = task->index_point[0];
  auto num_ranks = task->index_domain.get_volume();

  LEGATE_CHECK(task->futures.size() == 1);
  auto cpu_comm = task->futures[0].get_result<comm::coll::CollComm>();

  int device = -1;
  LEGATE_CHECK_CUDA(hipGetDevice(&device));

  /* Create communicator */
  cal_comm_t cal_comm = nullptr;
  cal_comm_create_params_t params;
  params.allgather    = allgather;
  params.req_test     = request_test;
  params.req_free     = request_free;
  params.data         = reinterpret_cast<void*>(cpu_comm);
  params.rank         = rank;
  params.nranks       = num_ranks;
  params.local_device = device;

  CHECK_CAL(cal_comm_create(params, &cal_comm));

  return cal_comm;
}

void finalize_cal(const Legion::Task* task,
                  const std::vector<Legion::PhysicalRegion>& /*regions*/,
                  Legion::Context context,
                  Legion::Runtime* runtime)
{
  legate::nvtx::Range auto_range{"core::comm::cal::finalize"};

  legate::detail::show_progress(task, context, runtime);

  LEGATE_CHECK(task->futures.size() == 1);
  auto comm = task->futures[0].get_result<cal_comm_t>();
  CHECK_CAL(cal_comm_destroy(comm));
}

}  // namespace

void register_tasks(const detail::Library* core_library)
{
  const auto runtime = Legion::Runtime::get_runtime();
  runtime->register_task_variant<cal_comm_t, init_cal>(
    detail::make_registrar(core_library,
                           LEGATE_CORE_INIT_CAL_TASK_ID,
                           "core::comm::cal::init",
                           Processor::TOC_PROC,
                           true),
    LEGATE_GPU_VARIANT);
  runtime->register_task_variant<finalize_cal>(
    detail::make_registrar(core_library,
                           LEGATE_CORE_FINALIZE_CAL_TASK_ID,
                           "core::comm::cal::finalize",
                           Processor::TOC_PROC,
                           true),
    LEGATE_GPU_VARIANT);
}

void register_factory(const detail::Library* core_library)
{
  auto* comm_mgr = detail::Runtime::get_runtime()->communicator_manager();
  comm_mgr->register_factory("cal", std::make_unique<Factory>(core_library));
}

}  // namespace legate::comm::cal
