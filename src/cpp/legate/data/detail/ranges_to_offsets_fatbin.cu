#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <legate/utilities/typedefs.h>

#include <cstddef>

namespace {

__device__ __forceinline__ std::size_t global_tid_1d()
{
  return static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
}

}  // namespace

extern "C" __global__ void legate_ranges_to_offsets_kernel(
  std::size_t ranges_volume,
  legate::Point<1> ranges_lo,
  legate::AccessorWO<std::int32_t, 1> offsets_acc,
  legate::AccessorRO<legate::Rect<1>, 1> ranges_acc)
{
  if (const auto tid = global_tid_1d(); tid < ranges_volume) {
    const auto p = ranges_lo + tid;

    offsets_acc[p] = ranges_acc[p].lo[0] - ranges_acc[ranges_lo].lo[0];
  }
}
